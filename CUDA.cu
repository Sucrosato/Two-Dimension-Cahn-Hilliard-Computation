﻿#include <iostream>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

using namespace std;

#define Nx 128
#define Ny 128
#define h 1.0
#define nstep 50000
#define nprint 1000
#define dt 0.01
#define c0 0.4
#define mob 1.0
#define grad_coef 0.4
#define noise 0.02

void init(double c[][Ny])
{
    //初始化
    for (int i = 0; i < Nx; i++)
        for (int j = 0; j < Ny; j++)
            c[i][j] = c0 + ((rand() % 10001 - 5000) / 10000.0) * noise;
    return;
}

__device__ double free_energy(double c)
{
    //计算自由能
    double result;
    double A = 1.0;
    result = A * (2.0 * c * pow(1 - c, 2) - 2 * pow(c, 2) * (1 - c));
    return result;
}

void print(double c[][Ny], int step)
{
    char filename[100] = {};
    sprintf(filename, "Data\\time_%d.dat", step);
    FILE* file = fopen(filename, "w");
    fprintf(file,
        "VARIABLES=\"x\",\"y\",\"c\"\n"
        "ZONE I=%d,J=%d,F=POINT\n",
        Nx, Ny);

    for (int i = 0; i < Nx; i++)   //输入当前进程分块的序参量的信息
        for (int j = 0; j < Ny; j++)
            fprintf(file, "%d    %d    %lf\n", i, j, c[i][j]);
    fclose(file);
    return;
}

__device__ double laplace(int x, int y, double a[][Ny])
{
    //计算矩阵a在(x,y)处的拉普拉斯算子
    double lap;
    int r, l, u, d;
    r = (x + 1) % Nx;
    l = (x - 1 + Nx) % Nx;
    u = (y - 1 + Ny) % Ny;
    d = (y + 1) % Ny;
    double n, e, s, w, ne, nw, se, sw, center;
    center = a[x][y];
    n = a[x][u];
    e = a[r][y];
    s = a[x][d];
    w = a[l][y];
    ne = a[r][u];
    nw = a[l][u];
    se = a[r][d];
    sw = a[l][d];
    lap = (((n + e + s + w) * 4 + (ne + nw + se + sw)) - 20 * center) / (6 * h * h);
    return lap;
}

__global__ void cCal(double c[][Ny], double f[][Ny])
{
    int tix = threadIdx.x;
    int bix = blockIdx.x;
    f[bix][tix] = free_energy(c[bix][tix]) - grad_coef * laplace(bix, tix, c);
    return;
}

__global__ void fCal(double c[][Ny], double f[][Ny])
{
    int tix = threadIdx.x;
    int bix = blockIdx.x;
    c[bix][tix] = c[bix][tix] + dt * mob * laplace(bix, tix, f);
    return;
}

int main(int argc, char* argv[])
{
    double c[Nx][Ny];
    clock_t start = clock();
    
    init(c);
    double (*d_c)[Ny], (*d_f)[Ny];
    hipMalloc(&d_c, sizeof(double) * Nx * Ny);
    hipMalloc(&d_f, sizeof(double) * Nx * Ny);
    hipMemcpy(d_c, c, sizeof(double) * Nx * Ny, hipMemcpyHostToDevice);
    dim3 grid(Nx, 1, 1), block(Ny, 1, 1);

    for (int step = 0; step <= nstep; step++)
    {
        fCal<<<grid, block>>>(d_c, d_f);
        cCal<<<grid, block>>>(d_c, d_f);
        hipMemcpy(c, d_c, sizeof(double) * Nx * Ny, hipMemcpyDeviceToHost);
        if(step%nprint==0)
        {
            hipMemcpy(c, d_c, sizeof(double) * Nx * Ny, hipMemcpyDeviceToHost);
            print(c, step);
        }
    }

    clock_t end = clock();
    double duration = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Total time: %.2fs\n", duration);
    hipFree(d_c);
    hipFree(d_f);
    return 0;
}

